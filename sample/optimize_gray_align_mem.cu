#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>

int div1(int M, int N) { 
	return ((M - 1) / N + 1);
}

#define KESTREL_KERNEL_LOOP(i, n)                                                                \
        for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);                         \
             i += blockDim.x * gridDim.x)

#define MAX_THREAD_IN_BLOCK (512)
#define KESTREL_KERNEL_CFG(total)                                                                \
        ((total + MAX_THREAD_IN_BLOCK - 1) / MAX_THREAD_IN_BLOCK), MAX_THREAD_IN_BLOCK

texture<unsigned char, 1, hipReadModeElementType> texture1_;
texture<unsigned char, 2, hipReadModeElementType> texture2_;

void __global__ gray_kernel1(unsigned char* data, float* output, int w, int h , int stride) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < w * h) {
		int _w = index % w;
		int _h = index / w;
		unsigned char* pixel = (unsigned char*)(data + (stride * _h  + _w * 3));
		output[index] = pixel[0] * 0.114 + pixel[1] * 0.587 + pixel[2] * 0.299;
	} 
}


__global__ void gray_kernel2(uint8_t *data, float *outdata, int32_t w, int32_t h, int32_t stride)
{
        KESTREL_KERNEL_LOOP(index, w * h)
        {
                int _h = index / w;
                int _w = index % w;
                const uint8_t *IMAGE = (uint8_t *)(data + stride * _h + 3 * sizeof(uint8_t) * _w);
                outdata[index] = IMAGE[0] * 0.114 + IMAGE[1] * 0.587 + IMAGE[2] * 0.299;
        }
}


__global__ void gray_kernel3(unsigned char* data, float* output, int w, int h, int stride) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < w && y < h) {
		unsigned char* pixel = data + (y * stride + x * 3);
		output[y * w + x] = pixel[0] * 0.114 + pixel[1] * 0.587 + pixel[2] * 0.299;
	}
}

__global__ void gray_kernel4(unsigned char* data, float* output, int w, int h, int stride) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		//unsigned char* pixel = tex1Dfetch(texture1_, stride * y + x * 3);
		output[y * w + x] = tex1Dfetch(texture1_, stride * y + x * 3) * 0.114 + tex1Dfetch(texture1_, stride * y + x * 3 + 1) * 0.587 + tex1Dfetch(texture1_, stride * y + x * 3 + 2) * 0.299;
		//output[y*w + x] = tex2D(texture1_, x, y) * 0.114 + tex2D(texture1_, x+1, y) * 0.587 + tex2D(texture1_, x+2, y) * 0.299; 
	}
}

__global__ void gray_kernel5(unsigned char* data, float* output, int w, int h, int stride) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int index = x * 3;
	if (x < w && y < h) {
		output[y * w + x] = tex2D(texture2_, index, y) * 0.114 + tex2D(texture2_, index + 1, y) * 0.587 + tex2D(texture2_, index + 2, y) * 0.299; 
	}
}

__global__ void gray_kernel6(unsigned char* data, float* output, int w, int h, int stride) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z;
	float scalar[] = {0.114f, 0.587f, 0.299f};
	int index = x * 3;
	if (x < w && y < h) {
		//output[y * w + x] = tex2D(texture2_, index, y) * 0.114 + tex2D(texture2_, index + 1, y) * 0.587 + tex2D(texture2_, index + 2, y) * 0.299; 
		output[y*w +x] += tex2D(texture2_, index + z, y) * scalar[z];
		//atomicAdd(output + (y*w + x), tex2D(texture2_, index + z, y) * scalar[z]);
	}
}


int main(int argc, char* argv[]) {
	
	const std::string filename(argv[1]);
	const int batch_size = atoi(argv[2]);
	std::cout << "batch size:" << batch_size << std::endl;
	cv::Mat image = cv::imread(filename);
	if (!image.isContinuous()) {
		std::cout << "read image fail." << std::endl;
		return -1;
	}
		

	unsigned char* d_align = nullptr;	
	unsigned char* d_image_input = nullptr;
	float* d_image_gray = nullptr;

	const int w = image.cols;
	const int h = image.rows;
	const int stride = image.step;
	const int channel = image.channels();
	printf("input dims [%d, %d, %d, %d].\n", channel, w, h, stride);

	hipMalloc((void**)&d_image_input, w * h * channel * sizeof(char));
	size_t pitch = 0;
	hipMallocPitch(&d_align, &pitch, stride, h);
	std::cout << "alloc pitch:" << pitch << std::endl;
	hipMalloc((void**)&d_image_gray, w * h * sizeof(float));
	hipMemcpy(d_image_input, image.data, w * h * channel * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy2D(d_align, pitch, image.data, stride, w*channel, h, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	const int threadsInBlock = 512;
	const int block_thread = 16;
	for (int i = 0; i < batch_size; ++i) {
		/*			
		dim3 grid(div1(w * h, threadsInBlock), 1, 1);
		dim3 block(threadsInBlock, 1, 1);
		//gray_kernel1<<<grid, block>>>(d_image_input, d_image_gray, w, h, stride);
		gray_kernel2<<<KESTREL_KERNEL_CFG(w * h), 0>>>(d_image_input, d_image_gray, w, h, stride);
		
		*/
		
		/*
		hipChannelFormatDesc channelDesc =
                             hipCreateChannelDesc<unsigned char>();	
				
		hipBindTexture(NULL, texture1_, d_image_input, pitch * h);
		dim3 block(block_thread, block_thread, 1);
		dim3 grid(div1(w, block_thread), div1(h, block_thread), 1);
		//gray_kernel3<<<grid, block>>>(d_align, d_image_gray, w, h, pitch);
		//gray_kernel4<<<grid, block>>>(d_image_input, d_image_gray, w, h, pitch);
		*/
		
		hipChannelFormatDesc channelDesc =
		     hipCreateChannelDesc<unsigned char>();	
		hipError_t err = hipBindTexture2D(0, texture2_, d_align, channelDesc, w*3, h, pitch);	
		dim3 block(block_thread, block_thread, 1);
		dim3 grid(div1(w, block_thread), div1(h, block_thread), 1);
		gray_kernel5<<<grid, block>>>(d_align, d_image_gray, w, h, pitch);
		
		/*
		hipError_t err = hipBindTexture2D(0, texture2_, d_image_input, channelDesc, w*3, h, stride);	
		std::cout << "bindtexture err:" << err << std::endl;
		dim3 block(block_thread, block_thread, 1);
		dim3 grid(div1(w, block_thread), div1(h, block_thread), 3);
		gray_kernel6<<<grid, block>>>(d_image_input, d_image_gray, w, h, stride);
		*/		
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float used_time = 0.0f;
	hipEventElapsedTime(&used_time, start, stop);

	int len = w * h;
	float *h_buffer = new float[len];
	hipMemcpy(h_buffer, d_image_gray, w * h * sizeof(float), hipMemcpyDeviceToHost);
	cv::Mat h_img(h, w, CV_32FC1, h_buffer);

	cv::Mat gray_img;
	h_img.convertTo(gray_img, CV_8U);
	cv::imshow("picture", gray_img);
	cv::waitKey();	

	std::cout << "cuda kernel run time:" << used_time << "ms" << std::endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_image_input);
	hipFree(d_image_gray);
	hipFree(d_align);
	return 0;
}
